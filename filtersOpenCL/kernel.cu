﻿
#include <chrono>
#include <CL/cl.h>
#include <iostream>
#include "filters.h"
#include "lodepng/lodepng.h"
#include "lodepng/lodepng.cpp"

using namespace std;

bool loadPNG(vector<unsigned char>& input, unsigned int& w, unsigned int& h, const string& fileName)
{
    return !lodepng::decode(input, w, h, fileName, LCT_RGB);
}

bool savePNG(vector<unsigned char>& input, unsigned int& w, unsigned int& h, const string& fileName)
{
    return !lodepng::encode(fileName, input, w, h, LCT_RGB);
}

int main()
{   
	vector<string> names = { "300x300", "400x400", "500x500", "600x600", "950x950", "2400x2400" };
	string format = ".png";

	vector<double> aveTimes(names.size(), 0.0);
	int iterations = 1000;

	fstream file("medFilterCL.txt");
	MedianFilter filter;

	for (int i = 0; i < names.size(); i++)
	{
		string nameFile = names[i];
		vector<unsigned char> img;
		unsigned int w, h;

		file << "Open file: " + nameFile + format + " ";
		cout << "Open file: " + nameFile + format + " ";

		if (loadPNG(img, w, h, "imageInput\\" + nameFile + format))
		{
			file << "success" << endl;
			file << "CalculationOfAverageTime: its-" << iterations << " -> ";
			cout << "success" << endl;
			cout << "CalculationOfAverageTime: its-" << iterations << " -> ";

			for (int j = 0; j < iterations; j++)
			{
				auto start = chrono::steady_clock::now();

				filter.processing(img, w, h, 16);

				auto end = chrono::steady_clock::now();
				double seconds = chrono::duration_cast<chrono::microseconds>(end - start).count() / 1000000.0;

				aveTimes[i] += seconds;
			}

			file << "aveTimes-" << aveTimes[i] / (double)iterations << " -> fullTime-" << aveTimes[i] << endl;
			cout << "aveTimes-" << aveTimes[i] / (double)iterations << " -> fullTime-" << aveTimes[i] << endl;
		}
		else
		{
			file << "fail" << endl;
			cout << "fail" << endl;
		}
		file << endl;
		cout << endl;
	}
    
    return 0;
}